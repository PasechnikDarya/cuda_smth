
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024 * 1024;

__global__ void kernel(float* arr) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926 * (float) i / (float) N;

  arr[i] = sinf(sqrtf(x));
}

int main() {
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float timerValueGRU;

  float *shared_arr;

  hipEventRecord(start, 0);

  hipMallocManaged((void**) &shared_arr, N * sizeof(float));

  kernel <<< N / 256, 256 >>> (shared_arr);

  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGRU, start, stop);

  printf("\n GPU computation time: %f ms\n", timerValueGRU);

  hipFree(shared_arr);

  return 0; 
}
