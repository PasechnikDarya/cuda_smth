
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024 * 1024;

__global__ void kernel(float* arr) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926 * (float) i / (float) N;

  arr[i] = sinf(sqrtf(x));
}

int main() {
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float timerValueGRU;

  float *arr, *cuda_arr;

  hipEventRecord(start, 0);

  arr = (float *) calloc (N, sizeof(float));
  // pinned memory
  // cudaHostAlloc((void **) &arr, N * sizeof(float), cudaHostAllocDefault);

  hipMalloc((void**) &cuda_arr, N * sizeof(float));

  kernel <<< N / 256, 256 >>> (cuda_arr);

  hipMemcpy(arr, cuda_arr, N * sizeof(float), hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGRU, start, stop);

  printf("\n GPU computation time: %f ms\n", timerValueGRU);

  free(arr);
  // cudaFreeHost(arr);
  hipFree(cuda_arr);

  return 0; 
}
