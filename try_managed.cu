
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024 * 1024;
__device__ __managed__ int managed_arr[N];


__global__ void kernel(void) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = 2.0f * 3.1415926 * (float) i / (float) N;

  managed_arr[i] = sinf(sqrtf(x));
}

int main() {
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float timerValueGRU;
  hipEventRecord(start, 0);

  kernel <<< N / 256, 256 >>> ();
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timerValueGRU, start, stop);

  printf("\n GPU computation time: %f ms\n", timerValueGRU);

  return 0; 
}
